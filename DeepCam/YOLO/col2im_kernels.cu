#include "hip/hip_runtime.h"
extern "C" {
#include "col2im.h"
#include "hip/hip_runtime.h"
}

// src: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
// You may also want to read: https://github.com/BVLC/caffe/blob/master/LICENSE

__global__ void col2im_gpu_kernel(const int n, const float* data_col,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
        const int height_col, const int width_col,
        float *data_im) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for(; index < n; index += blockDim.x*gridDim.x){
        float val = 0;
        int w = index % width + pad;
        int h = (index / width) % height + pad;
        int c = index / (width * height);
        // compute the start and end of the output
        int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
        int w_col_end = min(w / stride + 1, width_col);
        int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
        int h_col_end = min(h / stride + 1, height_col);
        // equivalent implementation
        int offset =
            (c * ksize * ksize + h * ksize + w) * height_col * width_col;
        int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
        int coeff_w_col = (1 - stride * height_col * width_col);
        for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
            for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
                val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
            }
        }
        data_im[index] = val;
    }
}

void col2im_ongpu(float *data_col,
        int channels, int height, int width,
        int ksize, int stride, int pad, float *data_im){
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    pad = pad ? ksize/2 : 0;
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    int num_kernels = channels * height * width;
    col2im_gpu_kernel<<<(num_kernels+BLOCK-1)/BLOCK,
        BLOCK>>>(
                num_kernels, data_col, height, width, ksize, pad,
                stride, height_col,
                width_col, data_im);
}

/*
   __global__ void col2im_kernel(float *data_col,
   int channels, int height, int width,
   int ksize, int stride, int pad, float *data_im)
   {

   int height_col = (height - ksize) / stride + 1;
   int width_col = (width - ksize) / stride + 1;
   if (pad){
   height_col = 1 + (height-1) / stride;
   width_col = 1 + (width-1) / stride;
   pad = ksize/2;
   }

   int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
   if(id >= channels*height*width) return;

   int index = id;
   int w = id%width + pad;
   id /= width;
   int h = id%height + pad;
   id /= height;
   int c = id%channels;

   int w_start = (w-ksize+stride)/stride;
   int w_end = w/stride + 1;

   int h_start = (h-ksize+stride)/stride;
   int h_end = h/stride + 1;

// int rows = channels * ksize * ksize;
// int cols = height_col*width_col;
int col_offset = (c*ksize*ksize + h * ksize + w)*height_col*width_col;
int h_coeff = (1-stride*ksize*height_col)*width_col;
int w_coeff = 1-stride*height_col*width_col;
float val = 0;
int h_col, w_col;
for(h_col = h_start; h_col < h_end; ++h_col){
for(w_col = w_start; w_col < w_end; ++w_col){
int col_index = col_offset +h_col*h_coeff + w_col*w_coeff;
float part = (w_col < 0 || h_col < 0 || h_col >= height_col || w_col >= width_col) ? 0 : data_col[col_index];
val += part;
}
}
data_im[index] = val;
}


extern "C" void col2im_ongpu(float *data_col,
int channels,  int height,  int width,
int ksize,  int stride,  int pad, float *data_im)
{

size_t n = channels*height*width;

col2im_kernel<<<cuda_gridsize(n), BLOCK>>>(data_col, channels, height, width, ksize, stride, pad, data_im);
check_error(hipPeekAtLastError());
}
 */
